#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <regex.h>
#include <hip/hip_runtime.h>
#include "../src/includes/env.h"
#include "../src/includes/logic_data.cu"
#include "./llist_test.cu"
#include "./search_data_test.cu"


#define SIZE 3

// NVIDIA P106-100
// Main function
int main()
{
  LogicData* logicData = logicInit();
  printf("123 <> \n");
  printf("count %d \n", logicData->count);

  logicAddInt(logicData, "FIELD", L_EQUAL, 10);
  logicAddMiddle(logicData, L_AND);
  logicAddInt(logicData, "P1", L_EQUAL, 33);
  logicAddMiddle(logicData, L_OR);
  logicAddString(logicData, "P2", L_EQUAL, "char data");

  logicPrint(logicData);
  logicFree(logicData);

  llist_test();
  searchData_test();
  return 0;
}


