#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <regex.h>
#include <hip/hip_runtime.h>
#include "../src/includes/env.h"
#include "../src/includes/logic_data.cu"
#include "./llist_test.cu"
#include "./search_data_test.cu"
#include "./logic_data_test.cu"
#include "./json_search_data_test.cu"


#define SIZE 3

// NVIDIA P106-100
// Main function
int main()
{
  llist_test();
  searchData_test();
  logicData_test();
  json_search_data_test();
  return 0;
}


