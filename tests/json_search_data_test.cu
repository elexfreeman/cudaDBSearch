#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <regex.h>
#include <hip/hip_runtime.h>
#include "../src/includes/env.h"
#include "../src/includes/logic_data.cu"
#include <json-c/json.h>

#define SIZE 3
void doarray(json_object *obj);

void doit(json_object *obj)
{
  json_object_object_foreach(obj, key, val)
  {
    switch (json_object_get_type(val))
    {
    case json_type_array:
      printf("\n%s  \n\n", key);
      doarray(val);
      break;

    case json_type_object:
      printf("\n%s  \n\n", key);
      doit(val);
      break;

    default:
      printf("%s: %s\n", key, json_object_get_string(val));
    }
  }
}

void doarray(json_object *obj)
{
  int temp_n = json_object_array_length(obj);
  const char *str;
  for (int i = 0; i < temp_n; i++)
  {
    switch (json_object_get_type(json_object_array_get_idx(obj, i)))
    {
    case json_type_array:
      doarray(json_object_array_get_idx(obj, i));
      break;

    case json_type_object:
      doit(json_object_array_get_idx(obj, i));
      break;

    default:
      str= json_object_get_string(json_object_array_get_idx(obj, i));
      printf("The value at %i position is: %s\n", i, str);
    }
  }
}

void parceDataItem(struct json_object *obj)
{
  if(!obj) return;
  json_object_object_foreach(obj, key, val)
  {
    switch (json_object_get_type(val))
    {
    case json_type_array:
      printf("\n%s  \n\n", key);
//      doarray(val);
      break;

    case json_type_object:
      printf("\n%s  \n\n", key);
//     doit(val);
      break;

    default:
      printf("%s: %s\n", key, json_object_get_string(val));
    }
  }
}

// NVIDIA P106-100
// Main function
int json_search_data_test()
{

// Declare a variable to store a pointer to the file.

  char* filename = strdup("./src/data.json");
  FILE* file = fopen(filename, "r");
  if (file == NULL)
  {
    fprintf(stderr, "Failed to open file: %s\n", filename);
    return 1;
  }

  // Determine the file size
  fseek(file, 0, SEEK_END);
  long file_size = ftell(file);
  rewind(file);

  // Allocate memory for the file content
  char* content = (char*)malloc(file_size + 1);
  if (content == NULL)
  {
    fprintf(stderr, "Failed to allocate memory for file content.\n");
    fclose(file);
    return 1;
  }

  // Read the file content into memory
  size_t bytes_read = fread(content, 1, file_size, file);
  if (bytes_read != file_size)
  {
    fprintf(stderr, "Error reading file: %s\n", filename);
    free(content);
    fclose(file);
    return 1;
  }
  free(filename);

  // Null-terminate the content
  content[file_size] = '\0';

  // Print the file content
//  printf("File content:\n%s\n", content);


  json_object *root = json_tokener_parse(content);

  printf(" >>>>>>>>>>>>>>>>>>>>>> \n");
  doit(root);


  // Cleanup
  json_object_put(root);
  free(content);
  fclose(file);
  return 0;
}


