
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAX_LOGIC_SIZE 200
#define C_EQUAL '='
#define L_EQUAL 1
#define C_AND '&'
#define L_AND 2

#define VT_STRING 1
#define VT_INT 2
#define VT_FLOAT 3
#define VT_MIDDLE 0


typedef struct
{
  char* key;
  int logic;
  void* value;
  int valueType;
} LogicItem;

typedef struct
{
  LogicItem* data;
  int count;
} LogicData;

LogicData* logicInit()
{
  LogicData* logicData = (LogicData*)malloc(sizeof(LogicData));
  logicData->count = 0;
  logicData->data = (LogicItem*)malloc(sizeof(LogicItem)*MAX_LOGIC_SIZE);

  return logicData;
}

void logicFree(LogicData* logicData)
{

  if(logicData->count!=0)
  {
    for(int k=0; k< logicData->count; k++)
    {
      free(logicData->data[k].key);
      free(logicData->data[k].value);
    }
    free(logicData->data);

  }
  free(logicData);
}

void logicAddInt(LogicData* logicData, char* key, int logic, int value)
{
  int len =  strlen(key);
  logicData->data[logicData->count].key = (char*)malloc(sizeof(char)*len);
  memcpy(logicData->data[logicData->count].key, key, len);

  logicData->data[logicData->count].logic = logic;
  logicData->data[logicData->count].valueType = VT_INT;

  int* valueTmp = (int*)malloc(sizeof(int));
  *valueTmp = value;
  logicData->data[logicData->count].value = valueTmp;
  logicData->count++;
}

void logicAddFloat(LogicData* logicData, char* key, int logic, float value)
{
  int len =  strlen(key);
  logicData->data[logicData->count].key = (char*)malloc(sizeof(char)*len);
  memcpy(logicData->data[logicData->count].key, key, len);

  logicData->data[logicData->count].logic = logic;
  logicData->data[logicData->count].valueType = VT_INT;

  float* valueTmp =  (float*)malloc(sizeof(float));
  *valueTmp = value;
  logicData->data[logicData->count].value = valueTmp;
  logicData->count++;
}


void logicAddMiddle(LogicData* logicData, int logic)
{
  logicData->data[logicData->count].key = 0;
  logicData->data[logicData->count].logic = logic;
  logicData->data[logicData->count].valueType = VT_MIDDLE;
  logicData->data[logicData->count].value = 0;
  logicData->count++;
}

char getLogicChar(int logic)
{
  if(logic==L_EQUAL) return C_EQUAL;
  if(logic==L_AND) return C_AND;
  return ' ';
}

void logicPrint(LogicData* logicData)
{
  char* logicStr = (char*)malloc(sizeof(char)*MAX_LOGIC_SIZE);

  for(int k=0; k<logicData->count; k++)
  {
    if(logicData->data[k].valueType==VT_INT)
    {
      sprintf(logicStr, "%s %s %c %d ",
              logicStr,
              logicData->data[k].key,
              getLogicChar(logicData->data[k].logic),
              *((int*)logicData->data[k].value)
             );
    }
    if(logicData->data[k].valueType==VT_MIDDLE)
    {
      sprintf(logicStr, "%s %c ",
              logicStr,
              getLogicChar(logicData->data[k].logic)
             );
    }
  }
  printf("%s\n", logicStr);
  free(logicStr);
}

