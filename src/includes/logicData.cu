
#include <hip/hip_runtime.h>
#define MAX_LOGIC_SIZE 200
#define C_EQUAL "="
#define L_EQUAL 1
#define C_AND "&"
#define L_AND 2

#define VT_STRING 1
#define VT_INT 2
#define VT_FLOAT 3


typedef struct
{
  int key;
  int logic;
  void* value;
  int valueType;
} LogicItem;

typedef struct
{
  char* name;
  int valueType;
  int key;
} FieldItem;


typedef struct
{
  LogicItem* data;
  int count;
  FieldItem* fieldData;
  int fieldCount;
} LogicData;

LogicData* logicInit()
{
  LogicData* logicData = (LogicData*)malloc(sizeof(LogicData));
  logicData->count = 0;
  logicData->fieldCount = 0;
  logicData->data = (LogicItem*)malloc(sizeof(LogicItem)*MAX_LOGIC_SIZE);
  logicData->fieldData = (FieldItem*)malloc(sizeof(FieldItem)*MAX_LOGIC_SIZE);

  return logicData;
}

void logicFree(LogicData* logicData)
{
  if(logicData->fieldCount!=0)
  {
    for(int k=0; k< logicData->fieldCount; k++)
    {
      free(logicData->fieldData[k].name);
    }
    free(logicData->fieldData);
  }

  if(logicData->count!=0)
  {
    for(int k=0; k< logicData->count; k++)
    {
      free(logicData->data[k].value);
    }
    free(logicData->data);

  }
  free(logicData);
}

int logicGetFieldItemKey(LogicData* logicData, char* name)
{
  int out = -1;
  for(int k=0; k< logicData->fieldCount; k++)
  {
    if(strcmp(logicData->fieldData[k].name, name)==0)
    {
      out = k;
      break;
    }
  }
  return out;
}

void logicAddField(LogicData* logicData, char* fieldName, int valueType)
{
  int key = logicGetFieldItemKey(logicData, fieldName);

  if(key >=0) return;

  logicData->fieldData[logicData->fieldCount].key = logicData->fieldCount;
  logicData->fieldData[logicData->fieldCount].valueType = valueType;

  int len =  strlen(fieldName);
  logicData->fieldData[logicData->fieldCount].name = (char*)malloc(sizeof(char)*len);
  memcpy(logicData->fieldData[logicData->fieldCount].name, fieldName, len);

  logicData->fieldCount++;
}

void logicAddInt(LogicData* logicData, char* fieldName, int logic, int value)
{
  int key = logicGetFieldItemKey(logicData, fieldName);
  logicData->data[logicData->count].key = key;
  logicData->data[logicData->count].logic = logic;
  logicData->data[logicData->count].valueType = VT_INT;

  int* valueTmp = (int*)malloc(sizeof(int));
  *valueTmp = value;
  logicData->data[logicData->count].value = valueTmp;
  logicData->count++;
}

void logicAddFloat(LogicData logicData, int key, int logic, float value)
{
  logicData.data[logicData.count].key = key;
  logicData.data[logicData.count].logic = logic;
  logicData.data[logicData.count].valueType = VT_INT;

  float* valueTmp =  (float*)malloc(sizeof(float));
  *valueTmp = value;
  logicData.data[logicData.count].value = valueTmp;
  logicData.count++;
}


void logicAddMiddle(LogicData logicData, int logic)
{
  logicData.data[logicData.count].key = -1;
  logicData.data[logicData.count].logic = logic;
  logicData.data[logicData.count].valueType = -1;
  logicData.data[logicData.count].value = 0;
  logicData.count++;
}

