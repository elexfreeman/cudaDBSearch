#include "hip/hip_runtime.h"
#ifndef _SEARCH_DATA_CU
#define _SEARCH_DATA_CU

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "./env.h"
/*
product: [
  {
    price: 100,
    tags: ['supper', 'versal', 'hateyou'],
  },
  {
    price: 200,
    tags: ['live', 'in', 'my', 'dream'],
  },
  {
    heart: 1,
    tags: ['live', 'in', 'my', 'dream'],
    soul: 2,
  },
  {
    heart: 3,
    youmyheart: [1, 3, 7],
    voyje: 2.33,
  },
]
*/


/*
__global__ void mallocTest()
{
    char* ptr = (char*)malloc(123);
    printf(“Thread %d got pointer: %p\n”, threadIdx.x, ptr);
    free(ptr);
}

void main()
{
    // Set a heap size of 128 megabytes. Note that this must
    // be done before any kernel is launched.
    cudaThreadSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
    mallocTest<<<1, 5>>>();
    hipDeviceSynchronize();
}
*/

typedef struct
{
  int valueType;
  void* data;
} SearchDataItem;


#endif
