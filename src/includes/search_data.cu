#include "hip/hip_runtime.h"
#ifndef _SEARCH_DATA_CU
#define _SEARCH_DATA_CU

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "./env.h"
#include "./llist.cu"
/*
__global__ void mallocTest()
{
    char* ptr = (char*)malloc(123);
    printf(“Thread %d got pointer: %p\n”, threadIdx.x, ptr);
    free(ptr);
}

void main()
{
    // Set a heap size of 128 megabytes. Note that this must
    // be done before any kernel is launched.
    cudaThreadSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
    mallocTest<<<1, 5>>>();
    hipDeviceSynchronize();
}
*/
/*
SearchData* searchData : [
  {
    price: {
      valueType: VT_INT,
      data: 33,
    },
    tags: {
      valueType: VT_A_STRING,
      data:['supper', 'versal', 'hateyou'],
    }
  }
]
*/



typedef struct
{
  char* name;
  int valueType;
  void* data;
} SearchDataItem;


LList* searchDataInit()
{
  return createList();
}

void searchDataAdd(LList* searchData, SearchDataItem* item)
{
  addToList(searchData, (void*)item);
}


void searchDataAddInt(LList* searchData, char* name, int data)
{
  SearchDataItem* item = (SearchDataItem*)malloc(sizeof(SearchDataItem));
  item->data = (void*)malloc(sizeof(int));
  hipMemcpy(item->data, &data, sizeof(int), hipMemcpyHostToHost);
  item->name = name;
  item->valueType = VT_INT;

  addToList(searchData, (void*)item);
}

void searchDataAddFloat(LList* searchData, char* name, float data)
{
  SearchDataItem* item = (SearchDataItem*)malloc(sizeof(SearchDataItem));
  item->data = (void*)malloc(sizeof(float));
  hipMemcpy(item->data, &data, sizeof(float), hipMemcpyHostToHost);
  item->name = name;
  item->valueType = VT_FLOAT;

  addToList(searchData, (void*)item);
}

void searchDataAddStr(LList* searchData, char* name, char* data)
{
  SearchDataItem* item = (SearchDataItem*)malloc(sizeof(SearchDataItem));

  item->data = data;
  item->name = name;
  item->valueType = VT_STRING;

  addToList(searchData, (void*)item);
}

void searchDataPrint(LList* searchData)
{
  LNode* currentNode = searchData->head;
  SearchDataItem* item = 0;
  int i = 0;
  while (currentNode != NULL)
  {
    LNode* nextNode = currentNode->next;
    SearchDataItem* item = (SearchDataItem*)currentNode->data;
    if(item->valueType == VT_INT)
    {
      printf("%d:%s=%d \n", i, item->name, *((int*)item->data));
    }
    if(item->valueType == VT_STRING)
    {
      printf("%d:%s=%s \n", i, item->name, (char*)item->data);
    }
    if(item->valueType == VT_FLOAT)
    {
      printf("%d:%s=%f \n", i, item->name, *((float*)item->data));
    }

    i++;
    currentNode = nextNode;
  }
}

void searchDataFree(LList* searchData)
{
  LNode* currentNode = searchData->head;
  while (currentNode != NULL)
  {
    LNode* next_node = currentNode->next;
    SearchDataItem* item = (SearchDataItem*)currentNode->data;
    if(item == NULL)
    {
      continue;
    }

    if(item->name != NULL)
    {
      printf("free name \n");
      free(item->name);
    }
    if(item->data != NULL)
    {
      printf("free data \n");
      free(item->data);
    }

    currentNode = next_node;
  }
  freeList(searchData);
}

#endif
