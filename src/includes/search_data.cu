#include "hip/hip_runtime.h"
#ifndef _SEARCH_DATA_CU
#define _SEARCH_DATA_CU

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "./env.h"
/*
__global__ void mallocTest()
{
    char* ptr = (char*)malloc(123);
    printf(“Thread %d got pointer: %p\n”, threadIdx.x, ptr);
    free(ptr);
}

void main()
{
    // Set a heap size of 128 megabytes. Note that this must
    // be done before any kernel is launched.
    cudaThreadSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
    mallocTest<<<1, 5>>>();
    hipDeviceSynchronize();
}
*/
/*
SearchData* searchData : [
  {
    price: {
      valueType: 'int',
      data: 33,
    },
    tags: ['supper', 'versal', 'hateyou'],
  },
  {
    price: 200,
    tags: ['live', 'in', 'my', 'dream'],
  },
  {
    heart: 1,
    tags: ['live', 'in', 'my', 'dream'],
    soul: 2,
  },
  {
    heart: 3,
    youmyheart: [1, 3, 7],
    voyje: 2.33,
  },
]
*/



typedef struct
{
  char* name;
  int valueType;
  void* data;
} SearchDataItem;


typedef struct
{
  SearchDataItem* dataItem;
  int count;
} SearchData;


#endif
