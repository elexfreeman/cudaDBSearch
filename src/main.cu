#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <regex.h>
#include <hip/hip_runtime.h>
#include "./includes/env.h"
#include "./includes/logic_data.cu"
#include <json-c/json.h>

#define SIZE 3
void doarray(json_object *obj);

void doit(json_object *obj)
{
  json_object_object_foreach(obj, key, val)
  {
    switch (json_object_get_type(val))
    {
    case json_type_array:
      printf("\n%s  \n\n", key);
      doarray(val);
      break;

    case json_type_object:
      printf("\n%s  \n\n", key);
      doit(val);
      break;

    default:
      printf("%s: %s\n", key, json_object_get_string(val));
    }
  }
}

void doarray(json_object *obj)
{
  int temp_n = json_object_array_length(obj);
  const char *str;
  for (int i = 0; i < temp_n; i++)
  {
    switch (json_object_get_type(json_object_array_get_idx(obj, i)))
    {
    case json_type_array:
      doarray(json_object_array_get_idx(obj, i));
      break;

    case json_type_object:
      doit(json_object_array_get_idx(obj, i));
      break;

    default:
      str= json_object_get_string(json_object_array_get_idx(obj, i));
      printf("The value at %i position is: %s\n", i, str);
    }
  }
}

void parceDataItem(struct json_object *obj)
{
  if(!obj) return;
  json_object_object_foreach(obj, key, val)
  {
    switch (json_object_get_type(val))
    {
    case json_type_array:
      printf("\n%s  \n\n", key);
//      doarray(val);
      break;

    case json_type_object:
      printf("\n%s  \n\n", key);
//     doit(val);
      break;

    default:
      printf("%s: %s\n", key, json_object_get_string(val));
    }
  }
}

// NVIDIA P106-100
// Main function
int main()
{
  LogicData* logicData = logicInit();
  printf("123 <> \n");
  printf("count %d \n", logicData->count);

  logicAddInt(logicData, "FIELD", L_EQUAL, 10);
  logicAddMiddle(logicData, L_AND);
  logicAddInt(logicData, "P1", L_EQUAL, 33);
  logicAddMiddle(logicData, L_OR);
  logicAddString(logicData, "P2", L_EQUAL, "char data");

  logicPrint(logicData);
  logicFree(logicData);

  const char *str;


// Declare a variable to store a pointer to the file.

  char* filename ="./src/data.json";
  FILE* file = fopen(filename, "r");
  if (file == NULL)
  {
    fprintf(stderr, "Failed to open file: %s\n", filename);
    return 1;
  }

  // Determine the file size
  fseek(file, 0, SEEK_END);
  long file_size = ftell(file);
  rewind(file);

  // Allocate memory for the file content
  char* content = (char*)malloc(file_size + 1);
  if (content == NULL)
  {
    fprintf(stderr, "Failed to allocate memory for file content.\n");
    fclose(file);
    return 1;
  }

  // Read the file content into memory
  size_t bytes_read = fread(content, 1, file_size, file);
  if (bytes_read != file_size)
  {
    fprintf(stderr, "Error reading file: %s\n", filename);
    free(content);
    fclose(file);
    return 1;
  }

  // Null-terminate the content
  content[file_size] = '\0';

  // Print the file content
//  printf("File content:\n%s\n", content);


  json_object *root = json_tokener_parse(content);
  printf(" >>>>>>>>>>>>>>>>>>>>>> \n");
  doit(root);

 // printf("The json representation:\n\n%s\n\n", json_object_to_json_string_ext(root, JSON_C_TO_STRING_PRETTY));
  int n = json_object_array_length(root);
  for (int i=0; i<n; i++)
  {
    str= json_object_get_string(json_object_array_get_idx(root, i));
//    enum json_type type = json_object_get_type(json_object_array_get_idx(root, i));
//    switch (type)
//    {
//    case json_type_null:
//      printf("json_type_null \n");
//      break;
//    case json_type_boolean:
//      printf("json_type_boolean \n");
//      break;
//    case json_type_double:
//      printf("json_type_double \n");
//      break;
//    case json_type_int:
//      printf("json_type_int \n");
//      break;
//    case json_type_object:
//      printf("json_type_object \n");
//      break;
//    case json_type_array:
//      printf("json_type_array \n");
//      break;
//    case json_type_string:
//      printf("json_type_string \n");
//      break;
//    }
//    printf("The value at %i position is: %s\n", i, str);
  }

  json_object_put(root);

  // Cleanup
  free(content);
  fclose(file);
  return 0;
}

