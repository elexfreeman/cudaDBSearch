#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define SIZE 3

typedef struct
{
  int size;
  void* p_array;
} GpuArray;


// Define a struct to store data
typedef struct
{
  int size;
  int* array;
} Array;


// Define a kernel to add two arrays
__global__ void add_arrays(GpuArray* d_data, int* d_out)
{
  int thread_id = threadIdx.x;
//  int size = (d_data[thread_id].p_array)[0].size;
//  d_out[thread_id] = d_data[thread_id].size;
  d_out[thread_id] = ((int*)d_data[thread_id].p_array)[0];
}

GpuArray initDevArray(GpuArray data)
{

  //https://forums.developer.nvidia.com/t/dynamic-array-inside-struct/10455/6
  GpuArray d_data;
  d_data.size = data.size;
  d_data.p_array = 0;

  hipMalloc(&d_data.p_array, sizeof(int)*data.size);
  hipMemcpy(d_data.p_array, data.p_array, sizeof(int)*data.size, hipMemcpyHostToDevice);

  return d_data;
}

void exmplePtoP()
{
  int** p_k = (int**)malloc(sizeof(int*));
  int*  k = (int*)malloc(sizeof(int)*SIZE);
  for(int i=0; i<SIZE; i++)
  {
    k[i] = i+3;
  }
  *p_k = k;

  for(int i=0; i<SIZE; i++)
  {
    printf(">>> k = %d, add = %p \n",(*p_k)[i], &p_k[i]);
  }
}


// Main function
int main()
{

  GpuArray* h_dataList = (GpuArray*)malloc(SIZE*sizeof(GpuArray));

  for(int i=0; i<SIZE; i++)
  {
    h_dataList[i].size = SIZE;
    h_dataList[i].p_array = (int*)malloc(sizeof(int)*SIZE);
    for(int j=0; j<SIZE; j++)
    {
      ((int*)h_dataList[i].p_array)[j]  = 5;
      printf(">>> j=%d, k = %d, add = %p \n",
             i,
             ((int*)h_dataList[i].p_array)[j],
             &((int*)h_dataList[i].p_array)[j]);
    }
  }

  hipSetDevice(0);

  GpuArray* h_dataListWithDevData = (GpuArray*)malloc(SIZE*sizeof(GpuArray));
  for(int k=0; k< SIZE; k++)
  {
    h_dataListWithDevData[k]  = initDevArray(h_dataList[k]);
  }

  GpuArray* d_data = 0;
  hipMalloc(&d_data, sizeof(GpuArray)*SIZE);
  hipMemcpy(d_data, h_dataListWithDevData, sizeof(GpuArray)*SIZE, hipMemcpyHostToDevice);
  printf("dev = %p \n", d_data );

  int* h_out = (int*)malloc(SIZE*sizeof(int));
  int* d_out = 0;
  hipMalloc(&d_out, sizeof(int)*SIZE);
  printf("dev_out = %p \n", d_out );

  add_arrays <<<1, SIZE >>>(d_data, d_out);
  hipDeviceSynchronize();

  // Copy the arrays from the device back to the host
  hipMemcpy(h_out, d_out, SIZE* sizeof(int), hipMemcpyDeviceToHost);

  for(int k=0; k< SIZE; k++)
  {
    printf("out = %d \n", h_out[k] );
  }

  for(int k=0; k< SIZE; k++)
  {
    hipFree(h_dataListWithDevData[k].p_array);
    free(h_dataList[k].p_array);
  }

  hipFree(d_data);
  hipFree(d_out);

  free(h_out);
  free(h_dataListWithDevData);
  free(h_dataList);

  return 0;
}

