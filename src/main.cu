#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <regex.h>
#include <hip/hip_runtime.h>
#include "./includes/logicData.cu"

#define SIZE 3




// Main function
int main()
{
  LogicData* logicData = logicInit();
  printf("123 <> \n");
  printf("count %d \n", logicData->count);

  logicAddInt(logicData, "FIELD", L_EQUAL, 10);
  logicAddMiddle(logicData, L_AND);
  logicAddInt(logicData, "P1", L_EQUAL, 33);

  logicPrint(logicData);
  logicFree(logicData);

  return 0;
}

