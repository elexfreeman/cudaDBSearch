#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <regex.h>
#include <hip/hip_runtime.h>
#include "./includes/env.h"
#include "./includes/logic_data.cu"
#include <json-c/json.h>

#define SIZE 3

// NVIDIA P106-100
// Main function
int main()
{
  LogicData* logicData = logicInit();
  printf("123 <> \n");
  printf("count %d \n", logicData->count);

  logicAddInt(logicData, "FIELD", L_EQUAL, 10);
  logicAddMiddle(logicData, L_AND);
  logicAddInt(logicData, "P1", L_EQUAL, 33);
  logicAddMiddle(logicData, L_OR);
  logicAddString(logicData, "P2", L_EQUAL, "char data");

  logicPrint(logicData);
  logicFree(logicData);

  const char *str;


// Declare a variable to store a pointer to the file.

  char* filename ="./src/logic.json";
  FILE* file = fopen(filename, "r");
  if (file == NULL)
  {
    fprintf(stderr, "Failed to open file: %s\n", filename);
    return 1;
  }

  // Determine the file size
  fseek(file, 0, SEEK_END);
  long file_size = ftell(file);
  rewind(file);

  // Allocate memory for the file content
  char* content = (char*)malloc(file_size + 1);
  if (content == NULL)
  {
    fprintf(stderr, "Failed to allocate memory for file content.\n");
    fclose(file);
    return 1;
  }

  // Read the file content into memory
  size_t bytes_read = fread(content, 1, file_size, file);
  if (bytes_read != file_size)
  {
    fprintf(stderr, "Error reading file: %s\n", filename);
    free(content);
    fclose(file);
    return 1;
  }

  // Null-terminate the content
  content[file_size] = '\0';

  // Print the file content
  printf("File content:\n%s\n", content);


  json_object *root = json_tokener_parse(content);

  printf("The json representation:\n\n%s\n\n", json_object_to_json_string_ext(root, JSON_C_TO_STRING_PRETTY));

  int n = json_object_array_length(root);
  for (int i=0; i<n; i++)
  {
    str= json_object_get_string(json_object_array_get_idx(root, i));
    printf("The value at %i position is: %s\n", i, str);
  }

  json_object_put(root);

  // Cleanup
  free(content);
  fclose(file);
  return 0;
}

