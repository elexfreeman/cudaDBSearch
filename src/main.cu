#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <regex.h>
#include <hip/hip_runtime.h>
#include "./includes/env.h"
#include "./includes/logic_data.cu"

#define SIZE 3

// NVIDIA P106-100
// Main function
int main()
{
  LogicData* logicData = logicInit();
  printf("123 <> \n");
  printf("count %d \n", logicData->count);

  logicAddInt(logicData, "FIELD", L_EQUAL, 10);
  logicAddMiddle(logicData, L_AND);
  logicAddInt(logicData, "P1", L_EQUAL, 33);
  logicAddMiddle(logicData, L_OR);
  logicAddString(logicData, "P2", L_EQUAL, "char data");

  logicPrint(logicData);
  logicFree(logicData);

  return 0;
}

