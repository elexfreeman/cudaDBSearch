#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <regex.h>
#include <hip/hip_runtime.h>
#include "./includes/logicData.cu"

#define SIZE 3




// Main function
int main()
{
  LogicData* logicData = logicInit();
  printf("123 <> \n");
  printf("count %d \n", logicData->count);

  logicAddInt(logicData, "FIELD", L_EQUAL, 10);
  printf("count %d \n", logicData->count);


  logicFree(logicData);

  return 0;
}

