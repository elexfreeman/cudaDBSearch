#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define SIZE 3

typedef struct
{
  int size;
  int** data;
} GpuArray;


// Define a struct to store data
typedef struct
{
	int size;
	int* array;
} Array;


// Define a kernel to add two arrays
__global__ void add_arrays(Array* d_arrays, int num_arrays)
{

	// Get the thread ID
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

	// Check if the thread is within the bounds of the array
	if (thread_id < num_arrays)
	{
		// Add the two arrays
		for (int i = 0; i < d_arrays[thread_id].size; i++)
		{
			d_arrays[thread_id].array[i] = 5;
		}
	}
}


// Main function
int main()
{

  int** p_k = (int**)malloc(sizeof(int*));
  int*  k = (int*)malloc(sizeof(int)*SIZE);
  for(int i=0; i<SIZE; i++)
  {
    k[i] = i+3;
  }
  *p_k = k;

  for(int i=0; i<SIZE; i++)
  {
    printf(">>> k = %d, add = %p \n",(*p_k)[i], &p_k[i]);
  }

  GpuArray* dataList = (GpuArray*)malloc(SIZE*sizeof(GpuArray));
  for(int i=0; i<SIZE; i++)
  {
    (*dataList[i])->size = SIZE;
    dataList[i]->data = (int**)malloc(sizeof(int*));
    *dataList[i]->data = (int*)malloc(sizeof(int)*SIZE);
    for(int j=0; j<SIZE; j++)
    {
      (*dataList[i]->data)[j]  = j+1;
      printf(">>> k = %d, add = %p \n",(*dataList[i]->data)[j], &dataList[i]->data[j]);
    }
  }

}

