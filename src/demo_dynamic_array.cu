#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define SIZE 3

typedef struct
{
  int size;
  int* p_array;
} GpuArray;


// Define a struct to store data
typedef struct
{
  int size;
  int* array;
} Array;


// Define a kernel to add two arrays
__global__ void add_arrays(GpuArray* d_data, int* d_out)
{
  int thread_id = threadIdx.x;
//  int size = (d_data[thread_id].p_array)[0].size;
//  d_out[thread_id] = d_data[thread_id].size;
  d_out[thread_id] = d_data[thread_id].p_array[0];
}

GpuArray initDevArray(GpuArray data)
{

  //https://forums.developer.nvidia.com/t/dynamic-array-inside-struct/10455/6
  GpuArray d_data;
  d_data.size = data.size;
  d_data.p_array = 0;

  //array
  hipMalloc(&d_data.p_array, sizeof(int)*data.size);
//  printf("malloc \n");
  hipMemcpy(d_data.p_array, data.p_array, sizeof(int)*data.size, hipMemcpyHostToDevice);
  printf("malloc %p \n", d_data.p_array);
  // var
//  GpuArray* d_data = 0;
//  cudaMalloc(&d_data, sizeof(GpuArray));
//  cudaMemcpy(d_data, &h_data, sizeof(GpuArray), cudaMemcpyHostToDevice);

  return d_data;
}


// Main function
int main()
{

  int** p_k = (int**)malloc(sizeof(int*));
  int*  k = (int*)malloc(sizeof(int)*SIZE);
  for(int i=0; i<SIZE; i++)
  {
    k[i] = i+3;
  }
  *p_k = k;

  for(int i=0; i<SIZE; i++)
  {
    printf(">>> k = %d, add = %p \n",(*p_k)[i], &p_k[i]);
  }

  GpuArray* h_dataList = (GpuArray*)malloc(SIZE*sizeof(GpuArray));

  for(int i=0; i<SIZE; i++)
  {
    h_dataList[i].size = SIZE;
    h_dataList[i].p_array = (int*)malloc(sizeof(int)*SIZE);
    for(int j=0; j<SIZE; j++)
    {
      h_dataList[i].p_array[j]  = 5;
      printf(">>> j=%d, k = %d, add = %p \n",i,h_dataList[i].p_array[j], &h_dataList[i].p_array[j]);
    }
  }

  hipSetDevice(0);

  GpuArray* h_dataListWithDevData = (GpuArray*)malloc(SIZE*sizeof(GpuArray));
  for(int k=0; k< SIZE; k++)
  {
    GpuArray temp = initDevArray(h_dataList[k]);
    h_dataListWithDevData[k].size  = temp.size;
    printf(" size = %d ", temp.size );
    h_dataListWithDevData[k].p_array  = temp.p_array;
  }

  GpuArray* d_data = 0;
  printf("dev = %d \n", d_data );
  hipMalloc(&d_data, sizeof(GpuArray)*SIZE);
  hipMemcpy(d_data, h_dataListWithDevData, sizeof(GpuArray)*SIZE, hipMemcpyHostToDevice);
  printf("dev = %p \n", d_data );

  int* h_out = (int*)malloc(SIZE*sizeof(int));
  int* d_out = 0;
  hipMalloc(&d_out, sizeof(int)*SIZE);
  printf("dev_out = %p \n", d_out );

  add_arrays <<<1, SIZE >>>(d_data, d_out);
  hipDeviceSynchronize();

  // Copy the arrays from the device back to the host
  hipMemcpy(h_out, d_out, SIZE* sizeof(int), hipMemcpyDeviceToHost);

  for(int k=0; k< SIZE; k++)
  {
    printf("out = %d \n", h_out[k] );
  }

  return 0;


  // Initialize the number of arrays
  printf("+++++ a \n");
  // Allocate memory for the arrays on the host
  Array* h_arrays = (Array*)malloc(sizeof(Array) * SIZE);

  printf("b \n");
  // Initialize the arrays on the host
  for (int i = 0; i < SIZE; i++)
  {
    printf("c");
    h_arrays[i].size = SIZE;
    h_arrays[i].array = (int*)malloc(sizeof(int) * h_arrays[i].size);
    for (int j = 0; j < h_arrays[i].size; j++)
    {
      h_arrays[i].array[j] = 2;
    }
    printf("0");
    int* d_array = 0;
    // Allocate memory for the arrays on the device
    hipMalloc(&d_array, sizeof(Array) * SIZE);
    printf("1");
    // Copy the arrays from the host to the device
    hipMemcpy(d_array, h_arrays[i].array, sizeof(Array) * SIZE, hipMemcpyHostToDevice);
    printf("2");
    free(h_arrays[i].array);
    printf("3");
    h_arrays[i].array = d_array;
    printf("4");
  }

  // Allocate memory for the arrays on the device
  Array* d_arrays = 0;
  hipMalloc(&d_arrays, sizeof(Array) * SIZE);


  // Copy the arrays from the host to the device
  hipMemcpy(d_arrays, h_arrays, sizeof(Array) * SIZE, hipMemcpyHostToDevice);

  // Launch the kernel
//  add_arrays << <1024, 1024 >> >(d_arrays, SIZE);

  // Copy the arrays from the device back to the host
  hipMemcpy(h_arrays, d_arrays, sizeof(Array) * SIZE, hipMemcpyDeviceToHost);

  int* tmp = 0;
  for (int i = 0; i < SIZE; i++)
  {
    tmp = (int*)malloc(sizeof(int)* SIZE);
    hipMemcpy(h_arrays[i].array, tmp, sizeof(Array) * SIZE, hipMemcpyDeviceToHost);
    for (int j = 0; j < SIZE; j++)
    {
      printf("Array %d, element %d: %d\n", i, j, tmp[j]);
    }
    hipFree(h_arrays[i].array);

    h_arrays[i].array = (int*)malloc(sizeof(int)* SIZE);
    hipMemcpy(h_arrays[i].array, tmp, SIZE, hipMemcpyHostToHost );
    free(tmp);
  }

//  // Print the arrays
//  for (int i = 0; i < SIZE; i++)
//  {
//    for (int j = 0; j < h_arrays[i].size; j++)
//    {
//      printf("Array %d, element %d: %d\n", i, j, h_arrays[i].array[j]);
//    }
//  }


  // Free the memory on the host and device
  for (int i = 0; i < SIZE; i++)
  {
    free(h_arrays[i].array);
  }

  free(h_arrays);
  hipFree(d_arrays);


  // Return success
  return 0;
}


